#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello(){

  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main(){

  hello<<<2, 2>>>();
  // Kernel launches are asynchronous to the host thread, meaning the host thread will not wait for completion of kernel before 
  // proceeding to the next line of code.
  // Calling "hipDeviceSynchronize" synchronizes the host thread to kernels.
  hipDeviceSynchronize();
  return 0;
}l